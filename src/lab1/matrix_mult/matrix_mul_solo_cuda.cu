#include "hip/hip_runtime.h"
#include <stdio.h>
#include "matrix_mul.h"

// Thread block size
#define BLOCK_SIZE 16 

// Forward declaration of the device multiplication function
__global__ void Muld(float*A, float*B, int wA, int wB, float*C)
{
	//Asumiré que las dimensiones de C son múltiplos de BLOCK_SIZE
	int i = blockIdx.y * blockDim.y + threadIdx.y,
			j = blockIdx.x * blockDim.x + threadIdx.x;
	C[i*wB + j] = 0;
	for (int k = 0; k < wA; k++)
		C[i*wB + j] += A[i*wA + k] * B[k*wB + j];
}

// Host multiplication function
// Compute C = A * B
// hA is the height of A
// wA is the width of A
// wB is the width of B


void Mul___(float* A, float* B, int hA, int wA, int wB, float* C)
{
	int size;

	// Load A and B to the device
	float* Ad;
	size = hA * wA * sizeof(float);
	hipMalloc((void**)&Ad, size);
	hipMemcpy(Ad, A, size, hipMemcpyHostToDevice);
	float* Bd;
	size = wA * wB * sizeof(float);
	hipMalloc((void**)&Bd, size);
	hipMemcpy(Bd, B, size, hipMemcpyHostToDevice);

	// Allocate C on the device
	float* Cd;
	size = hA * wB * sizeof(float);
	hipMalloc((void**)&Cd, size);

	// Compute the execution configuration assuming
	// the matrix dimensions are multiples of BLOCK_SIZE
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(wB / dimBlock.x, hA / dimBlock.y);

	// Launch the device computation
	Muld<<<dimGrid, dimBlock>>>(Ad, Bd, wA, wB, Cd);

	// Read C from the device
	hipMemcpy(C, Cd, size, hipMemcpyDeviceToHost);

	// Free device memory
	hipFree(Ad);
	hipFree(Bd);
	hipFree(Cd);
}

#if 0
#endif
